/*
    Compilation: 
    
    nvcc -arch=sm_60 atomic-01.cu
    
    Execution example:
    
    $ CUDA_VISIBLE_DEVICES=0 ./a.out 
    atomic-01.cu 
    5000000 
    
    $ CUDA_VISIBLE_DEVICES=0,1 ./a.out 
    atomic-01.cu 
    5000269
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <vector>
using namespace std;

#define CU_SAFE(stmt) \
  do { \
    hipError_t status = stmt; \
    if (status != hipSuccess) { \
       printf("ERROR: %s in %s:%d.\n", hipGetErrorString(status), \
       __FILE__, __LINE__); \
       exit( 1 );  \
    } \
  } while (0)

#define WS 32   /* Warp size */
#define AllLanes (0xffffffff)

typedef long long unsigned u64;

/* = Managed ( Unified ) =================== */

__managed__ int itask_m = 0;

__managed__ u64 total_count_m = 0ULL;

/* ========================================= */

__global__  void Kernel( u64* v_d, int ntask, int nv );

void Count( )
{
  int ntask = 5000000;
  int nv = 50 * WS;

  // Multi Device Handling ----------------------

  int ndev;
  CU_SAFE( hipGetDeviceCount( &ndev ) );

  u64* v_d[ ndev ]; 

  for( int idev = 0 ; idev < ndev; idev ++ ) {

    CU_SAFE( hipSetDevice( idev ) );
    hipDeviceProp_t prop;
    CU_SAFE( hipGetDeviceProperties( &prop, idev ) );

    int nblocks = prop.multiProcessorCount;

    CU_SAFE( hipMalloc( v_d + idev, sizeof(u64) * nv *( 1 + nblocks ) ) );
    /* contents of this array don't matter. */

    Kernel <<< nblocks, WS >>> ( v_d[ idev ], ntask, nv );
  }

  for( int idev = 0 ; idev < ndev; idev ++ ) {

    CU_SAFE( hipSetDevice( idev ) );
    CU_SAFE( hipFree( v_d[ idev ] ) );  
  }

  return; 
}

__device__ int vcopy( u64* vp, int nv, u64* vpn ) 
{
   assert( ( nv % WS ) == 0 );
   int j = 0;
   for( int i = 0; i < nv; i += WS ) {
      vpn[ j + threadIdx.x ] = vp[ i + threadIdx. x ];
      j += WS;
   }
   return j;
}

__global__ void Kernel( u64* v_d, int ntask, int nv )
{
  u64* vb = v_d + nv * ( blockIdx.x + 1 );

  u64 count = 0ULL;

  for(   ; /* i_task < ntask */; /* i_task ++ */ ) {

     int itask;
     if( threadIdx.x == 0 ) {
        itask = atomicAdd_system( &itask_m, 1 );
     }
     itask = __shfl_sync( AllLanes, itask, 0 ); 

     if( itask >= ntask ) break;

     vcopy( v_d, nv, vb ); // just wasting time.
     vcopy( v_d, nv, vb );
     vcopy( v_d, nv, vb );
     vcopy( v_d, nv, vb );
     vcopy( v_d, nv, vb );
     vcopy( v_d, nv, vb );
     vcopy( v_d, nv, vb );
     vcopy( v_d, nv, vb );
     vcopy( v_d, nv, vb );
     vcopy( v_d, nv, vb );

     count ++;
  }

  if( threadIdx.x == 0 ) 
  {
     atomicAdd_system( &total_count_m, count );
  }
}

int main( int argc, char* argv[] )
{
   fprintf( stderr, "%s\n", __FILE__ );

   total_count_m = 0ULL; // initialize managed variable
   itask_m = 0; 

   Count( );

   printf( " %15llu \n", total_count_m ); 

   return 0;
}

